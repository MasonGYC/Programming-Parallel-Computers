/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
#include <cmath>
#include <vector>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

static inline void check(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b)
{
    return (a + b - 1) / b;
}

static inline int roundup(int a, int b)
{
    return divup(a, b) * b;
}

__global__ void myppkernel(float *dataGPU, int ny, int nx, int nn)
{
    // pad ny to be nn
    // nn: multiple of 64
    int ja = threadIdx.x; // col % 64
    int i = blockIdx.y;   // row, 1 block 1 row

    for (int jb = 0; jb < nx; jb += 64)
    {
        int j = jb + ja;
        if (j < nx && i >= ny)
        {
            dataGPU[j + i * nx] = 0;
        }
    }
}

__global__ void myNormkernel(float *data, float *normedGPU, int ny, int nx)
{
    // First normalize the input rows so that each row has the arithmetic mean of 0
    // Then normalize the input rows so that for each row the sum of the squares of the elements is 1

    int ja = threadIdx.x; // col % 64
    int i = blockIdx.y;   // row, 1 block 1 row

    int y = i * 64 + ja;
    if (y < ny)
    {
        // get average of this row
        float ave = 0;
        for (int x = 0; x < nx; x++)
        {
            ave += data[x + y * nx];
        }
        ave = ave / nx;

        // get std of this row
        float std = 0;
        for (int x = 0; x < nx; x++)
        {
            std += pow((data[x + y * nx] - ave), 2);
        }
        std = sqrt(std);

        // fill in X of this row
        for (int x = 0; x < nx; x++)
        {
            normedGPU[x + y * nx] = (data[x + y * nx] - ave) / std;
        }
    }
}

__global__ void mykernel(float *resGPU, float *dataGPU, int ny, int nx, int nn)
{
    int thread_x = threadIdx.x; // x: col
    int thread_y = threadIdx.y; // y: row
    int block_x = blockIdx.x;
    int block_y = blockIdx.y;

    if (block_x < block_y)
    {
        for (int ib = 0; ib < 8; ++ib)
        {
            for (int jb = 0; jb < 8; ++jb)
            {
                int i = block_y * 64 + ib * 8 + thread_y;
                int j = block_x * 64 + jb * 8 + thread_x;
                if (i < ny && j < ny)
                {
                    resGPU[j + i * ny] = 0;
                }
            }
        }
    }
    else
    {
        float v[8][8] = {0}; // store result of row i * j
        const int size = 6;

        // loop all elements in a row
        for (int k = 0; k < nx; k += size)
        {
            float x[8][size];
            float y[8][size];

            for (int ib = 0; ib < 8; ++ib)
            {
                int i = block_y * 64 + ib * 8 + thread_y;
                for (int m = 0; m < size; m++)
                {
                    y[ib][m] = (k + m < nx) ? dataGPU[k + m + i * nx] : 0;
                }
            }
            for (int jb = 0; jb < 8; ++jb)
            {
                int j = block_x * 64 + jb * 8 + thread_x;
                for (int m = 0; m < size; m++)
                {
                    x[jb][m] = (k + m < nx) ? dataGPU[k + m + j * nx] : 0;
                }
            }
            for (int ib = 0; ib < 8; ++ib)
            {
                for (int jb = 0; jb < 8; ++jb)
                {
                    for (int m = 0; m < size; m++)
                    {
                        v[ib][jb] += y[ib][m] * x[jb][m];
                    }
                }
            }
        }
        for (int ib = 0; ib < 8; ++ib)
        {
            for (int jb = 0; jb < 8; ++jb)
            {
                int i = block_y * 64 + ib * 8 + thread_y;
                int j = block_x * 64 + jb * 8 + thread_x;
                if (i < ny && j < ny)
                {
                    resGPU[j + i * ny] = v[ib][jb];
                }
            }
        }
    }
}

void correlate(int ny, int nx, const float *data, float *result)
{

    int nn = roundup(ny, 64);
    int original_size = ny * nx * sizeof(float);

    float *rawGPU = NULL;
    CHECK(hipMalloc((void **)&rawGPU, original_size));
    float *dataGPU = NULL;
    CHECK(hipMalloc((void **)&dataGPU, nn * nx * sizeof(float)));
    float *resGPU = NULL;
    CHECK(hipMalloc((void **)&resGPU, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(rawGPU, data, original_size, hipMemcpyHostToDevice));

    // Run normalization kernel
    {
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, nn / 64);
        myNormkernel<<<dimGrid, dimBlock>>>(rawGPU, dataGPU, ny, nx);
        // CHECK(cudaDeviceSynchronize());
        CHECK(hipGetLastError());
    }

    // Run padding kernel
    {
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, nn);
        myppkernel<<<dimGrid, dimBlock>>>(dataGPU, ny, nx, nn);
        // CHECK(cudaDeviceSynchronize());
        CHECK(hipGetLastError());
    }

    // Run kernel
    {
        dim3 dimBlock(8, 8);
        dim3 dimGrid(nn / 64, nn / 64);
        mykernel<<<dimGrid, dimBlock>>>(resGPU, dataGPU, ny, nx, nn);
        // CHECK(cudaDeviceSynchronize());
        CHECK(hipGetLastError());
    }

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, resGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dataGPU));
    CHECK(hipFree(resGPU));
    CHECK(hipFree(rawGPU));
}