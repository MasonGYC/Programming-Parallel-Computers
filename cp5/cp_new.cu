/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
#include <cmath>
#include <vector>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

static inline void check(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b)
{
    return (a + b - 1) / b;
}

static inline int roundup(int a, int b)
{
    return divup(a, b) * b;
}

__global__ void myPadKernel(float *dataGPU, int ny, int nx, int nn_y, int nn_x)
{
    // pad ny to be nn
    // nn: multiple of 64
    int ja = threadIdx.x; // col % 64
    int i = blockIdx.y;   // row, 1 block 1 row

    for (int jb = 0; jb < nn_x; jb += 64)
    {
        int j = jb + ja;
        if (j < nx && i >= ny)
        {
            dataGPU[j + i * nx] = 0;
        }
    }
}

__global__ void myNormKernel(float *data, float *normedGPU, int ny, int nx, int nn_x)
{
    // First normalize the input rows so that each row has the arithmetic mean of 0
    // Then normalize the input rows so that for each row the sum of the squares of the elements is 1

    int ja = threadIdx.x; // col % 64
    int i = blockIdx.y;   // row, 1 block 1 row

    int y = i * 64 + ja;
    if (y < ny)
    {
        // get average of this row
        float ave = 0;
        for (int x = 0; x < nx; x++)
        {
            ave += data[x + y * nx];
        }
        ave = ave / nx;

        // get std of this row
        float std = 0;
        for (int x = 0; x < nx; x++)
        {
            std += pow((data[x + y * nx] - ave), 2);
        }
        std = sqrt(std);

        // fill in normedGPU of this row
        for (int x = 0; x < nx; x++)
        {
            normedGPU[x + y * nn_x] = (data[x + y * nx] - ave) / std;
        }
    }
}

__global__ void mykernel(float *resGPU, float *dataGPU, int ny, int nx, int nn_y, int nn_x)
{
    int thread_x = threadIdx.x; // x: row
    int thread_y = threadIdx.y; // y: col
    int block_x = blockIdx.x;
    int block_y = blockIdx.y;

    __shared__ float rowis[32][8];
    __shared__ float rowjs[32][8];

    if (block_x < block_y)
    {
        for (int ib = 0; ib < 8; ++ib)
        {
            for (int jb = 0; jb < 8; ++jb)
            {
                int i = block_y * 64 + ib * 8 + thread_y;
                int j = block_x * 64 + jb * 8 + thread_x;
                if (i < ny && j < ny)
                {
                    resGPU[j + i * ny] = 5;
                }
            }
        }
    }
    else
    {
        float v[8][8] = {0}; // store result of row i * j

        // store elements in shared memory
        // loop all elements in one row
        for (int k = 0; k < nn_x / 32; k++)
        {
            for (int cb = 0; cb < 4; cb++)
            {
                int col = k * 32 + cb * 8 + thread_y;
                int rowi = block_x * 8 + thread_x;
                int rowj = block_y * 8 + thread_x;
                rowis[col][thread_x] = dataGPU[nn_x * rowi + col];
                rowjs[col][thread_x] = dataGPU[nn_x * rowj + col];
            }
            __syncthreads();

#pragma unroll
            for (int c = 0; c < 32; c++)
            {
                float y[8];
                for (int jb = 0; jb < 8; ++jb)
                {
                    y[jb] = rowjs[c][jb];
                }
                for (int ib = 0; ib < 8; ++ib)
                {
                    float x = rowis[c][ib];
                    for (int jb = 0; jb < 8; ++jb)
                    {
                        v[ib][jb] += x * y[jb];
                    }
                }
            }

            __syncthreads();
        }

        // // loop all elements in a row
        // for (int k = 0; k < nx; k++)
        // {
        //     float x[8];
        //     float y[8];

        //     for (int ib = 0; ib < 8; ++ib)
        //     {
        //         int i = block_y * 64 + ib * 8 + thread_y;
        //         y[ib] = dataGPU[k + i * nx];
        //     }
        //     for (int jb = 0; jb < 8; ++jb)
        //     {
        //         int j = block_x * 64 + jb * 8 + thread_x;
        //         x[jb] = dataGPU[k + j * nx];
        //     }
        //     for (int ib = 0; ib < 8; ++ib)
        //     {
        //         for (int jb = 0; jb < 8; ++jb)
        //         {
        //             v[ib][jb] += y[ib] * x[jb];
        //         }
        //     }
        // }
        for (int ib = 0; ib < 8; ++ib)
        {
            for (int jb = 0; jb < 8; ++jb)
            {
                int i = block_y * 64 + ib * 8 + thread_y;
                int j = block_x * 64 + jb * 8 + thread_x;
                if (i < ny && j < ny)
                {
                    resGPU[j + i * ny] = v[ib][jb];
                }
            }
        }
    }
}

void correlate(int ny, int nx, const float *data, float *result)
{

    int nn_y = roundup(ny, 64);
    int nn_x = roundup(nx, 32);
    int original_size = ny * nx * sizeof(float);

    float *rawGPU = NULL;
    CHECK(hipMalloc((void **)&rawGPU, original_size));
    float *dataGPU = NULL;
    CHECK(hipMalloc((void **)&dataGPU, nn_y * nn_x * sizeof(float)));
    float *resGPU = NULL;
    CHECK(hipMalloc((void **)&resGPU, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(rawGPU, data, original_size, hipMemcpyHostToDevice));

    // Run normalization kernel
    {
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, nn_y / 64);
        myNormKernel<<<dimGrid, dimBlock>>>(rawGPU, dataGPU, ny, nx, nn_x);
        // CHECK(cudaDeviceSynchronize());
        CHECK(hipGetLastError());
    }

    // Run padding kernel
    {
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, nn_y);
        myPadKernel<<<dimGrid, dimBlock>>>(dataGPU, ny, nx, nn_y, nn_x);
        // CHECK(cudaDeviceSynchronize());
        CHECK(hipGetLastError());
    }

    // Run kernel
    {
        dim3 dimBlock(8, 8);
        dim3 dimGrid(nn_y / 64, nn_y / 64);
        mykernel<<<dimGrid, dimBlock>>>(resGPU, dataGPU, ny, nx, nn_y, nn_x);
        // CHECK(cudaDeviceSynchronize());
        CHECK(hipGetLastError());
    }

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, resGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dataGPU));
    CHECK(hipFree(resGPU));
    CHECK(hipFree(rawGPU));
}
