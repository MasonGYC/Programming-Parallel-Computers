/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
#include <cmath>
#include <vector>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

static inline void check(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b)
{
    return (a + b - 1) / b;
}

static inline int roundup(int a, int b)
{
    return divup(a, b) * b;
}

__global__ void mykernel(float *resGPU, float *dataGPU, int ny, int nx)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; // i: row, small ; blockDim.x = 16
    int j = threadIdx.y + blockIdx.y * blockDim.y; // j: col, big ; blockDim.y = 16
    // What if n is not a multiple of 16
    if (i >= ny || j >= ny)
        return;

    // Calculate the (upper triangle of the) matrix product Y = XXT.
    // store in result
    float temp = 0;
    for (int k = 0; k < nx; k++)
    {
        temp += dataGPU[k + i * nx] * dataGPU[k + j * nx];
    }
    resGPU[i * ny + j] = float(temp);
    // for (int y = 0; y < ny; y++)
    // {
    //     for (int x = y; x < ny; x++)
    //     {
    //         float temp = 0;
    //         for (int k = 0; k < nx; k++)
    //         {
    //             temp += dataGPU[k + y * nx] * dataGPU[k + x * nx];
    //         }
    //         resGPU[y * ny + x] = float(temp);
    //     }
    // }
}

void correlate(int ny, int nx, const float *data, float *result)
{
    // First normalize the input rows so that each row has the arithmetic mean of 0
    // Then normalize the input rows so that for each row the sum of the squares of the elements is 1

    float *X = (float *)malloc(sizeof(float) * ny * nx);
    for (int y = 0; y < ny; y++)
    {
        // get average of this row
        float ave = 0;
        for (int x = 0; x < nx; x++)
        {
            ave += data[x + y * nx];
        }
        ave = ave / nx;

        // get std of this row
        float std = 0;
        for (int x = 0; x < nx; x++)
        {
            std += pow((data[x + y * nx] - ave), 2);
        }
        std = sqrt(std);

        // fill in X of this row
        for (int x = 0; x < nx; x++)
        {
            X[x + y * nx] = (data[x + y * nx] - ave) / std;
        }
    }

    float *dataGPU = NULL;
    CHECK(hipMalloc((void **)&dataGPU, nx * ny * sizeof(float)));
    float *resGPU = NULL;
    CHECK(hipMalloc((void **)&resGPU, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(dataGPU, X, nx * ny * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
    mykernel<<<dimGrid, dimBlock>>>(resGPU, dataGPU, ny, nx);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, resGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dataGPU));
    CHECK(hipFree(resGPU));

    free(X);
}
